#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "sampling_cuda_kernel.h"


__device__ void __update(float *dists, int *dists_i, int idx1, int idx2) {
    const float v1 = dists[idx1], v2 = dists[idx2];
    const int i1 = dists_i[idx1], i2 = dists_i[idx2];
    dists[idx1] = max(v1, v2);
    dists_i[idx1] = v2 > v1 ? i2 : i1;
}

// input xyz: (n, 3), tmp: (b, n_max)
// output idx (m)
template <unsigned int block_size>
__global__ void furthestsampling_cuda_kernel(const float *xyz, const int *offset, const int *new_offset, float *tmp, int *idx)
{
    __shared__ float dists[block_size];
    __shared__ int dists_i[block_size];

    int bid = blockIdx.x;
    int start_n, end_n, start_m, end_m, old;
    if (bid == 0) {
        start_n = 0;
        end_n = offset[0];
        start_m = 0;
        end_m = new_offset[0];
        old = 0;
    }
    else {
        start_n = offset[bid - 1];
        end_n = offset[bid];
        start_m = new_offset[bid - 1];
        end_m = new_offset[bid];
        old = offset[bid - 1];
    }

    const int stride = block_size;
    int tid = threadIdx.x;
    if (tid == 0) idx[start_m] = start_n;

    __syncthreads();
    for (int j = start_m + 1; j < end_m; j++)
    {
        int besti = start_n;
        float best = -1;
        float x1 = xyz[old * 3 + 0];
        float y1 = xyz[old * 3 + 1];
        float z1 = xyz[old * 3 + 2];
        for (int k = start_n + tid; k < end_n; k += stride)
        {
            float x2 = xyz[k * 3 + 0];
            float y2 = xyz[k * 3 + 1];
            float z2 = xyz[k * 3 + 2];
            float d = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
            float d2 = min(d, tmp[k]);
            tmp[k] = d2;
            besti = d2 > best ? k : besti;
            best = d2 > best ? d2 : best;
        }
        dists[tid] = best;
        dists_i[tid] = besti;
        __syncthreads();

        if (block_size >= 1024) {
            if (tid < 512) {
            __update(dists, dists_i, tid, tid + 512);
            }
            __syncthreads();
        }
        if (block_size >= 512) {
            if (tid < 256) {
            __update(dists, dists_i, tid, tid + 256);
            }
            __syncthreads();
        }
        if (block_size >= 256) {
            if (tid < 128) {
            __update(dists, dists_i, tid, tid + 128);
            }
            __syncthreads();
        }
        if (block_size >= 128) {
            if (tid < 64) {
            __update(dists, dists_i, tid, tid + 64);
            }
            __syncthreads();
        }
        if (block_size >= 64) {
            if (tid < 32) {
            __update(dists, dists_i, tid, tid + 32);
            }
            __syncthreads();
        }
        if (block_size >= 32) {
            if (tid < 16) {
            __update(dists, dists_i, tid, tid + 16);
            }
            __syncthreads();
        }
        if (block_size >= 16) {
            if (tid < 8) {
            __update(dists, dists_i, tid, tid + 8);
            }
            __syncthreads();
        }
        if (block_size >= 8) {
            if (tid < 4) {
            __update(dists, dists_i, tid, tid + 4);
            }
            __syncthreads();
        }
        if (block_size >= 4) {
            if (tid < 2) {
            __update(dists, dists_i, tid, tid + 2);
            }
            __syncthreads();
        }
        if (block_size >= 2) {
            if (tid < 1) {
            __update(dists, dists_i, tid, tid + 1);
            }
            __syncthreads();
        }

        old = dists_i[0];
        if (tid == 0)
            idx[j] = old;
    }
}

void furthestsampling_cuda_launcher(int b, int n, const float *xyz, const int *offset, const int *new_offset, float *tmp, int *idx)
{   
	unsigned int n_threads = opt_n_threads(n);
	switch (n_threads) {
        case 1024:
            furthestsampling_cuda_kernel<1024><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 512:
            furthestsampling_cuda_kernel<512><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 256:
            furthestsampling_cuda_kernel<256><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 128:
            furthestsampling_cuda_kernel<128><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 64:
            furthestsampling_cuda_kernel<64><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 32:
            furthestsampling_cuda_kernel<32><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 16:
            furthestsampling_cuda_kernel<16><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 8:
            furthestsampling_cuda_kernel<8><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 4:
            furthestsampling_cuda_kernel<4><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 2:
            furthestsampling_cuda_kernel<2><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        case 1:
            furthestsampling_cuda_kernel<1><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
            break;
        default:
            furthestsampling_cuda_kernel<512><<<b, n_threads, 0>>>(xyz, offset, new_offset, tmp, idx);
    }
}
