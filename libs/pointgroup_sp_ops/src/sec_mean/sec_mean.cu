#include "hip/hip_runtime.h"
/*
Segment Operations (mean, max, min) (no bp)
Written by Li Jiang
All Rights Reserved 2020.
*/

#include <stdio.h>
#include <math.h>
#include "sec_mean.h"

/* ================================== sec_mean ================================== */
__global__ void sec_mean_cuda_(int nProposal, int C, float *inp, int *offsets, float *out){
    for(int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x){
        int start = offsets[p_id];
        int end = offsets[p_id + 1];

        float count = (float)(end - start);

        for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
            float mean = 0;
            for(int i = start; i < end; i++){
                mean += (inp[i * C + plane] / count);
            }
            out[p_id * C + plane] = mean;
        }
    }
}

//input: inp (N, C) float
//input: offsets (nProposal + 1) int
//output: out (nProposal, C) float
void sec_mean_cuda(int nProposal, int C, float *inp, int *offsets, float *out){
    sec_mean_cuda_<<<std::min(nProposal, (int)32768), std::min(C, (int)32)>>>(nProposal, C, inp, offsets, out);
}


/* ================================== sec_min ================================== */
__global__ void sec_min_cuda_(int nProposal, int C, float *inp, int *offsets, float *out){
    for(int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x){
        int start = offsets[p_id];
        int end = offsets[p_id + 1];

        for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
            float min_val = 1e50;
            for(int i = start; i < end; i++){
                if(inp[i * C + plane] < min_val){
                    min_val = inp[i * C + plane];
                }
            }
            out[p_id * C + plane] = min_val;
        }
    }
}

//input: inp (N, C) float
//input: offsets (nProposal + 1) int
//output: out (nProposal, C) float
void sec_min_cuda(int nProposal, int C, float *inp, int *offsets, float *out){
    sec_min_cuda_<<<std::min(nProposal, (int)32768), std::min(C, (int)32)>>>(nProposal, C, inp, offsets, out);
}


/* ================================== sec_max ================================== */
__global__ void sec_max_cuda_(int nProposal, int C, float *inp, int *offsets, float *out){
    for(int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x){
        int start = offsets[p_id];
        int end = offsets[p_id + 1];

        for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
            float max_val = -1e50;
            for(int i = start; i < end; i++){
                if(inp[i * C + plane] > max_val){
                    max_val = inp[i * C + plane];
                }
            }
            out[p_id * C + plane] = max_val;
        }
    }
}

//input: inp (N, C) float
//input: offsets (nProposal + 1) int
//output: out (nProposal, C) float
void sec_max_cuda(int nProposal, int C, float *inp, int *offsets, float *out){
    sec_max_cuda_<<<std::min(nProposal, (int)32768), std::min(C, (int)32)>>>(nProposal, C, inp, offsets, out);
}
